#include "hip/hip_runtime.h"
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/* DISTRIBUTED AND PARALLEL MATRIX-MATRIX PRODUCT WITH MPI AND CUDA          */
/*                                                                           */
/* File:         mmpmpicuda.cu                                               */
/* Author:       Alberto Pou Quirós (Github: bertini36)                      */ 
/* Description:  This program performs a matrix product (A * B = C)          */
/*               distributing the computation between multiple nodes         */
/*               with MPI technology and parallelizing the computation in    */
/*               every node with Nvidia CUDA technology                      */
/* Compilation:  nvcc -I/opt/mpi/bullxmpi/1.2.9.1/include                    */  
/*               -L/opt/mpi/bullxmpi/1.2.9.1/lib -lmpi -ldl -lm -lnuma       */
/*               -lrt -lnsl -lutil -lm -ldl mmpmpicuda.cu -o mmpmpicuda      */
/* Strategy:                                                                 */
/*                  Example 16x16 matrices with 4 nodes:                     */
/*                   _________________16________________                     */
/*                   |                                 |                     */
/*                   |               NODE 1            | 4                   */
/*                   |_________________________________|                     */
/*                   |                                 |                     */
/*                   |               NODE 2            | 4                   */
/*              C =  |_________________________________|    16               */ 
/*                   |                                 |                     */
/*                   |               NODE 3            | 4                   */
/*                   |_________________________________|                     */
/*                   |                                 |                     */ 
/*                   |               NODE 4            | 4                   */
/*                   |_________________________________|                     */
/*                                                                           */
/*                  Node 1 computes 4 rows of result matrix:                 */
/*                   __________________________________                      */
/*                   |                                 |                     */
/*                   |         4x16 CUDA block         |                     */
/*                   |_________________________________|                     */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <mpi.h>

#define N 1024

#define err(format, ...) do { fprintf(stderr, format, ##__VA_ARGS__); exit(1); } while (0)

struct timeval start_time, end_time;

inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        err("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

__global__ void matrixProduct(double *matrix_a, double *matrix_b, double *matrix_c, int width, int nrows, int my_rank) {
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x; 
    matrix_c[row * width + col] = 0;
    for (int k=0; k<width; k++) {
        matrix_c[row * width + col] += matrix_a[(row * width) + (my_rank * nrows * width) + k] * matrix_b[k * width + col];
    }
}

void initializeMatrices(double matrix_a[N][N], double matrix_b[N][N]) {
    int i, j;
    srand(time(NULL));
    for (i=0; i<N; i++) {
        for (j=0; j<N; j++) {
            matrix_a[i][j] = rand();
            matrix_b[i][j] = rand();
        }
    }
}

void showMatrices(double matrix_a[N][N], double matrix_b[N][N], double matrix_c[N][N]) {
    int i, j;
    srand(time(NULL));
    printf("***** MATRIX A ***** \n");
    for (i=0; i<N; i++) {
        for (j=0; j<N; j++) {
            (j % N == N-1) ? printf("%.1f \n", matrix_a[i][j]) : printf("%.1f,", matrix_a[i][j]);
        }
    }
    printf("***** MATRIX B ***** \n");
    for (i=0; i<N; i++) {
        for (j=0; j<N; j++) {
            (j % N == N-1) ? printf("%.1f \n", matrix_b[i][j]) : printf("%.1f,", matrix_b[i][j]);
        }
    }
    printf("***** RESULT MATRIX ***** \n");
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            (j % N == N-1) ? printf("%f \n", matrix_c[i][j]) : printf("%f,", matrix_c[i][j]);
        }
    }
}

int main(int argc, char *argv[]) {

    double A[N][N], B[N][N], C[N][N];
    double *d_a, *d_b, *d_c;
    int my_rank, comm_sz, from, to, nrows;
  
    // MPI initialization
    MPI_Init (&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);    // Process id 
    MPI_Comm_size(MPI_COMM_WORLD, &comm_sz);    // Number of processors 

    if (N % comm_sz != 0) {
        if (my_rank == 0) printf("Matrix size not divisible by number of processors \n");
        MPI_Finalize();
        exit(-1);
    }

    // Calculate interval lines to compute per node
    from = my_rank * N / comm_sz;
    to = (my_rank + 1) * N / comm_sz;
    nrows = to - from;

    if (my_rank == 0) { initializeMatrices(A, B); }

    // Send A y B to every node
    MPI_Bcast(A, N*N, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    MPI_Bcast(B, N*N, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    // Allocate memory in the device
    checkCuda(hipMalloc((void **) &d_a, N*N*sizeof(double)));
    checkCuda(hipMalloc((void **) &d_b, N*N*sizeof(double)));
    checkCuda(hipMalloc((void **) &d_c, (N*N/comm_sz)*sizeof(double)));

    // Copy the information in the device
    checkCuda(hipMemcpy(d_a, A, N*N*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, B, N*N*sizeof(double), hipMemcpyHostToDevice));

    // CUDA threads structure definition
    dim3 dimGrid(1);
    dim3 dimBlock(N, nrows);    

    MPI_Barrier(MPI_COMM_WORLD);
    if (my_rank == 0) { gettimeofday(&start_time, NULL); }

    // Kernel launch
    matrixProduct<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N, nrows, my_rank);
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipGetLastError());

    // Calculate compute time
    MPI_Barrier(MPI_COMM_WORLD);
    if (my_rank == 0) { 
        gettimeofday(&end_time, NULL);
        printf("Compute time: %.1f ms \n", (float) (end_time.tv_sec - start_time.tv_sec) * 1000 + (end_time.tv_usec - start_time.tv_usec) / 1000);
     }

    // Get results from device
    checkCuda(hipMemcpy(C[from], d_c, (nrows)*N*sizeof(double), hipMemcpyDeviceToHost));

    // Unify results from nodes
    MPI_Gather(C[from], N*N/comm_sz, MPI_DOUBLE, C, N*N/comm_sz, MPI_DOUBLE, 0, MPI_COMM_WORLD);

    // if (my_rank == 0)  { showMatrices(A, B, C); }

    checkCuda(hipFree(d_a));
    checkCuda(hipFree(d_b));
    checkCuda(hipFree(d_c));
    
    MPI_Finalize();
    return 0;

}